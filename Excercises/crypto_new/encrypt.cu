#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void encryptKernel(char* deviceDataIn, char* deviceDataOut) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    int key_enc = 1 ;
    if ((deviceDataIn[index]>='a' && deviceDataIn[index]<='z')||(deviceDataIn[index]>='A' && deviceDataIn[index]<='Z'))
    {
    deviceDataIn[index] = deviceDataIn[index];
    }
   // deviceDataIn[index] = ((deviceDataIn[index]>='A' && deviceDataIn[index]<='Z')?((deviceDataIn[index]>'Z')?(deviceDataIn[index]-'Z'+'A'-1):(deviceDataIn[index]+key)):(deviceDataIn[index]+key));
    deviceDataOut[index] = deviceDataIn[index];
}

__global__ void decryptKernel(char* deviceDataIn, char* deviceDataOut) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    //int dec_key = 1;
    if ((deviceDataIn[index]<='a' && deviceDataIn[index]<='z')||(deviceDataIn[index]>='A' && deviceDataIn[index]<='Z'))
    {
    deviceDataIn[index] = deviceDataIn[index];
    }
   
 //deviceDataIn[index]=deviceDataIn[index]-dec_key;
    deviceDataOut[index] = deviceDataIn[index];
}

int fileSize() {
  int size; 

  ifstream file ("original.data", ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.close();
  }
  else {
    cout << "Unable to open file";
    size = -1; 
  }
  return size; 
}

int readData(char *fileName, char *data) {

  streampos size;

  ifstream file (fileName, ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.seekg (0, ios::beg);
    file.read (data, size);
    file.close();

    cout << "The entire file content is in memory." << endl;
  }
  else cout << "Unable to open file" << endl;
  return 0;
}

int writeData(int size, char *fileName, char *data) {
  ofstream file (fileName, ios::out|ios::binary|ios::trunc);
  if (file.is_open())
  {
    file.write (data, size);
    file.close();

    cout << "The entire file content was written to file." << endl;
    return 0;
  }
  else cout << "Unable to open file";

  return -1; 
}

int EncryptSeq (int n, char* data_in, char* data_out) 
{  
  int i;
  int seq_key=1;
  int se_key=2;
  timer sequentialTime = timer("Sequential encryption");
  
  sequentialTime.start();
  for (i=0; i<n; i++) {
        if (i % 2 == 0){
       // cout<<"comes into the even encrypt"<<endl;
       // cout<<"i is "<<i<<endl;
       // cout<<"data is "<<data_in[i]<<endl;
        if (data_in[i] >='a' && data_in[i]<= 'z')
        {
        data_in[i]=data_in[i]+seq_key;
        if (data_in[i] > 'z'){
        data_in[i] = data_in[i]-'z'+'a'-1;
        }
        }
        else if (data_in[i] >= 'A' && data_in[i]<'Z')
        {
        data_in[i]=data_in[i]+seq_key;
	if(data_in[i] > 'Z'){
        data_in[i] = data_in[i]-'Z'+'A'-1;}
        }}
        else{
        cout<<"i is"<<i<<endl;
        cout<<"data is "<<data_in[i]<<endl;
        if (data_in[i] >='a' && data_in[i]<= 'z')
        {
        data_in[i] = data_in[i] + se_key;
        if (data_in[i] >'z'){
        data_in[i] = data_in[i]-'z'+ 'a' -1;
        }
        }
        else if (data_in[i] >= 'A' && data_in[i]<'Z')
        {
        data_in[i] = data_in[i]+se_key;
        if (data_in[i] >'Z'){
        data_in[i] = data_in[i]-'Z'+'A'-1;}
        }} 
        data_out[i]=data_in[i]; }
        
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Encryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;
  
  return 0; 
}

int DecryptSeq (int n, char* data_in, char* data_out)
{
  int i;
  timer sequentialTime = timer("Sequential decryption");
  int dec_seq_key = 1;
  int decs_key = 2;
  sequentialTime.start();
  for (i=0; i<n; i++) { 
	if(i % 2 == 0){
        cout<<"comes into the even";
        if (data_in[i] >='a' && data_in[i]<='z')
        {
        data_in[i]=data_in[i]-dec_seq_key;
        if (data_in[i] < 'a'){
        data_in[i] = data_in[i]+'z'-'a'+1;}
        }
        else if (data_in[i] >='A' && data_in[i]<='Z')
        {
        data_in[i]=data_in[i]-dec_seq_key;
        if (data_in[i]<'A'){ 
        data_in[i]= data_in[i]+'Z' - 'A' + 1;}
        }}
        
        else 
        {
        if (data_in[i] >='a' && data_in[i]<='z')
        {
        data_in[i] = data_in[i] - decs_key;
        if(data_in[i]<'a'){
        data_in[i] = data_in[i]+'z'-'a'+2;}
        }
        else if (data_in[i] >='A' && data_in[i]<='Z')
        {
        data_in[i]=data_in[i]-decs_key;
        }
        if (data_in[i]<'A'){
        data_in[i]=data_in[i]+'Z'-'A'+2;}
        }
        
	data_out[i]=data_in[i]; }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Decryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;

  return 0;
}


int EncryptCuda (int n, char* data_in, char* data_out) {
    int threadBlockSize = 128;

    // allocate the vectors on the GPU
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }
   // cout << deviceDataIn;
    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    encryptKernel<<<n/threadBlockSize+1, threadBlockSize>>>(deviceDataIn, deviceDataOut);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));

    cout << fixed << setprecision(6);
    cout << "Encrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Encrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int DecryptCuda (int n, char* data_in, char* data_out) {
    int threadBlockSize =128;

    // allocate the vectors on the GPU
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    decryptKernel<<<n/threadBlockSize+1, threadBlockSize>>>(deviceDataIn, deviceDataOut);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));

    cout << fixed << setprecision(6);
    cout << "Decrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Decrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int main(int argc, char* argv[]) {
    int n;

    n = fileSize();
    if (n == -1) {
	cout << "File not found! Exiting ... " << endl; 
	exit(0);
    }

    char* data_in = new char[n];
    char* data_out = new char[n];    
    readData("original.data", data_in); 

    cout << "Encrypting a file of " << n << " characters." << endl;

    EncryptSeq(n, data_in, data_out);
    writeData(n, "sequential.data", data_out);
    EncryptCuda(n, data_in, data_out);
    writeData(n, "cuda.data", data_out);  

    readData("cuda.data", data_in);
   // readData("sequential.data",data_in);
   cout << "Decrypting a file of " << n << "characters" << endl;
    DecryptSeq(n, data_in, data_out);
    writeData(n, "sequential_decrypted.data", data_out);
    DecryptCuda(n, data_in, data_out); 
    writeData(n, "recovered.data", data_out); 
 
    delete[] data_in;
    delete[] data_out;
    
    return 0;
}
